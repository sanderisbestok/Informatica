#include "hip/hip_runtime.h"
/*
 * simulate.c
 * Sander Hansen 10995080
 * Bas van Berckel 10343725
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "simulate.hpp"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << ": " << result << endl;
        exit(1);
    }
}


__global__ void waveStepKernel(double* deviceCur, double* deviceOld, double* deviceNext, int i_max) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if(!index || index >= i_max - 1) {
        deviceNext[index] = 0;
    } else {
        deviceNext[index] = (2 * deviceCur[index]) - deviceOld[index] + (0.15 *
                            (deviceCur[index - 1] - (2 * deviceCur[index] -
                            deviceCur[index + 1])));
    }
}


/*
 * Executes the entire simulation.
 *
 *
 * i_max: how many data points are on a single wave
 * t_max: how many iterations the simulation should run
 * num_threads: how many threads to use
 * old_array: array of size i_max filled with data for t-1
 * current_array: array of size i_max filled with data for t
 * next_array: array of size i_max. You should fill this with t+1
 */

double *simulate(const int i_max, const int t_max, const int threadBlockSize,
        double *old_array, double *current_array, double *next_array)
{

    // allocate the vectors on the GPU
    double* deviceOld = NULL;
    checkCudaCall(hipMalloc((void **) &deviceOld, i_max * sizeof(double)));
    if (deviceOld == NULL) {
        cout << "could not allocate memory!" << endl;
        return NULL;
    }
    double* deviceCur = NULL;
    checkCudaCall(hipMalloc((void **) &deviceCur, i_max * sizeof(double)));
    if (deviceCur == NULL) {
        checkCudaCall(hipFree(deviceOld));
        cout << "could not allocate memory!" << endl;
        return NULL;
    }
    double* deviceNext = NULL;
    checkCudaCall(hipMalloc((void **) &deviceNext, i_max * sizeof(double)));
    if (deviceNext == NULL) {
        checkCudaCall(hipFree(deviceOld));
        checkCudaCall(hipFree(deviceCur));
        cout << "could not allocate memory!" << endl;
        return NULL;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceOld, old_array, i_max*sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceCur, current_array, i_max*sizeof(double), hipMemcpyHostToDevice));

	double *temp;

	int t;
    hipEventRecord(start, 0);
	for (t = 0; t < t_max; t++) {
		waveStepKernel<<<i_max/threadBlockSize, threadBlockSize>>>(deviceCur, deviceOld, deviceNext, i_max);
        checkCudaCall(hipGetLastError());
        hipDeviceSynchronize();
        /* Swap buffers */
        temp = deviceOld;
        deviceOld = deviceCur;
        deviceCur = deviceNext;
        deviceNext = temp;
	}
    hipEventRecord(stop, 0);

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    checkCudaCall(hipMemcpy(current_array, deviceCur, i_max * sizeof(double), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceOld));
    checkCudaCall(hipFree(deviceCur));
    checkCudaCall(hipFree(deviceNext));

    cout << "Wave simulation took " << elapsedTime << " milliseconds with blocksize " << threadBlockSize << endl;

    return current_array;
}
